#include "hip/hip_runtime.h"
#include<torch/extension.h>

template<typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n>=feats.size(0) || f>=feats.size(2)) return;

    // point -1~1
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;
    
    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;
    feat_interp[n][f] = (1-u)*(a*feats[n][0][f] +
                               b*feats[n][1][f] +
                               c*feats[n][2][f] +
                               d*feats[n][3][f]) + 
                            u*(a*feats[n][4][f] +
                               b*feats[n][5][f] +
                               c*feats[n][6][f] +
                               d*feats[n][7][f]);
}
torch::Tensor trilinear_fw_cu(torch::Tensor features, torch::Tensor points){
    const int N = features.size(0), F = features.size(2);
    torch::Tensor interpolated_features = torch::zeros({N,F},  features.options());

    const dim3 threads(16, 16); // total 256 threads
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y - 1)/ threads.y);

    // launch kernel
    AT_DISPATCH_FLOATING_TYPES(features.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            features.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            interpolated_features.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return interpolated_features;
}