#include<torch/extension.h>

torch::Tensor trilinear_fw_cu(torch::Tensor features, torch::Tensor points){
    const int N = features.size(0), F = features.size(2);
    torch::Tensor interpolated_features = torch::zeros({N,F},  features.options());

    const dim3 threads(16, 16); // total 256 threads
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y - 1)/ threads.y);

    // launch kernel
    AT_DISPATCH_FLOATING_TYPES(features.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            feats.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            feat_interp.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return interpolated_features;
}