#include "hip/hip_runtime.h"
#include<torch/extension.h>

template<typename scalar_t>
__global__ void trilinear_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictPtrTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictPtrTraits, size_t> feat_interp
){

}
torch::Tensor trilinear_fw_cu(torch::Tensor features, torch::Tensor points){
    const int N = features.size(0), F = features.size(2);
    torch::Tensor interpolated_features = torch::zeros({N,F},  features.options());

    const dim3 threads(16, 16); // total 256 threads
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y - 1)/ threads.y);

    // launch kernel
    AT_DISPATCH_FLOATING_TYPES(features.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_kernel<scalar_t><<<blocks, threads>>>(
            features.packed_accessor<scalar_t, 3, torch::RestrictPtrTraits, size_t>(),
            points.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>(),
            interpolated_features.packed_accessor<scalar_t, 2, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return interpolated_features;
}